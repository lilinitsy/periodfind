#include "hip/hip_runtime.h"
// Copyright 2020 California Institute of Technology. All rights reserved.
// Use of this source code is governed by a BSD-style license that can be
// found in the LICENSE file.
// Author: Ethan Jaszewski

#include "ce.h"

#include <algorithm>
#include <iostream>

#include "hip/hip_runtime.h"
#include "math.h"

#include "errchk.cuh"

//
// Simple ConditionalEntropy Function Definitions
//

ConditionalEntropy::ConditionalEntropy(size_t n_phase,
									   size_t n_mag,
									   size_t p_overlap,
									   size_t m_overlap)
{
	// Just set number of bins
	num_phase_bins = n_phase;
	num_mag_bins   = n_mag;

	// Just set the overlap
	num_phase_overlap = p_overlap;
	num_mag_overlap   = m_overlap;

	// Calculate bin size accordingly
	phase_bin_size = 1.0 / static_cast<float>(n_phase);
	mag_bin_size   = 1.0 / static_cast<float>(n_mag);
}

__host__ __device__ size_t ConditionalEntropy::NumBins() const
{
	return num_phase_bins * num_mag_bins;
}

__host__ __device__ size_t ConditionalEntropy::NumPhaseBins() const
{
	return num_phase_bins;
}

__host__ __device__ size_t ConditionalEntropy::NumMagBins() const
{
	return num_mag_bins;
}

__host__ __device__ size_t ConditionalEntropy::NumPhaseBinOverlap() const
{
	return num_phase_overlap;
}

__host__ __device__ size_t ConditionalEntropy::NumMagBinOverlap() const
{
	return num_mag_overlap;
}

__host__ __device__ float ConditionalEntropy::PhaseBinSize() const
{
	return phase_bin_size;
}

__host__ __device__ float ConditionalEntropy::MagBinSize() const
{
	return mag_bin_size;
}

__host__ __device__ size_t ConditionalEntropy::PhaseBin(float phase_val) const
{
	return static_cast<size_t>(phase_val / phase_bin_size);
}

__host__ __device__ size_t ConditionalEntropy::MagBin(float mag_val) const
{
	return static_cast<size_t>(mag_val / mag_bin_size);
}

__host__ __device__ size_t ConditionalEntropy::BinIndex(size_t phase_bin,
														size_t mag_bin) const
{
	return phase_bin * num_mag_bins + mag_bin;
}

//
// CUDA Kernels
//

/**
 * Folds and bins the input data across all trial periods and time derivatives.
 *
 * This kernel takes in a time-series of paired times and magnitudes, folding
 * the times according to the given trial periods and time derivatives,
 * outputting a series of histograms into global memory.
 *
 * Each block computes a histogram of the full data series for a given period
 * and period time derivative. As such, the x-dimension of the grid should match
 * the number of trial periods, and the y-dimension of the grid should match the
 * number of trial period time derivatives.
 *
 * Internally, the kernel uses shared memory atomics with a 32-bit integer based
 * histogram, which requires a total of 4 * Histogram Size bytes of shared
 * memory. Due to the use of shared atomics, this kernel will perform poorly on
 * pre-Maxwell GPUs.
 *
 * Note: All arrays must be device-allocated
 *
 * @param times light curve datapoint times
 * @param mags light curve datapoint magnitudes
 * @param periods list of trial periods
 * @param period_dts list of trial period time derivatives
 * @param h_params histogram parameters
 * @param hists array of output histograms
 */
__global__ void FoldBinKernel(const float *__restrict__ times,
							  const float *__restrict__ mags,
							  const size_t length,
							  const float *__restrict__ periods,
							  const float *__restrict__ period_dts,
							  const ConditionalEntropy h_params,
							  float *__restrict__ hists)
{
	// Histogram which this block will produce.
	const size_t block_id          = blockIdx.x * gridDim.y + blockIdx.y;
	float *__restrict__ block_hist = &hists[h_params.NumBins() * block_id];

	// Period and period time derivative for this block.
	const float period    = periods[blockIdx.x];
	const float period_dt = period_dts[blockIdx.y];

	// Time derivative correction factor.
	const float pdt_corr = (period_dt / period) / 2;

	// Shared memory histogram for this thread.
	extern __shared__ uint32_t sh_hist[];

	// Zero the shared memory for this block
	for(size_t i = threadIdx.x; i < h_params.NumBins(); i += blockDim.x)
	{
		sh_hist[i] = 0;
	}

	__syncthreads();

	float i_part; // Only used for modff.

	// Accumulate into this thread's histogram (as many points as needed),
	// simultaneously computing the folded time value
	for(size_t idx = threadIdx.x; idx < length; idx += blockDim.x)
	{
		float t      = times[idx];
		float t_corr = t - pdt_corr * t * t;
		float folded = fabsf(modff(t_corr / period, &i_part));

		size_t phase_bin = h_params.PhaseBin(folded);
		size_t mag_bin   = h_params.MagBin(mags[idx]);

		for(size_t i = 0; i < h_params.NumPhaseBinOverlap(); i++)
		{
			for(size_t j = 0; j < h_params.NumMagBinOverlap(); j++)
			{
				size_t idx =
					h_params.BinIndex((phase_bin + i) % h_params.NumPhaseBins(), (mag_bin + j) % h_params.NumMagBins());
				atomicAdd(&sh_hist[idx], 1);
			}
		}
	}

	__syncthreads();

	size_t div = length * h_params.NumPhaseBinOverlap() * h_params.NumMagBinOverlap();

	// Copy the block's histogram into global memory
	for(size_t i = threadIdx.x; i < h_params.NumBins(); i += blockDim.x)
	{
		block_hist[i] = static_cast<float>(sh_hist[i]) / static_cast<float>(div);
	}
}

/**
 * Computes the conditional entropy for the input histograms.
 *
 * This kernel takes in an arbitrarily long list of histograms with a given set
 * of parameters and computes the conditional entropy for each histogram,
 * outputting a series of values into an array.
 *
 * Internally, each thread is responsible for first computing the conditional
 * entropy of one phase bin of the input (disregarding histogram boundaries),
 * then the values for each thread are accumulated directly into global memory
 * to avoid potential inter-block conflicts. This copies the histogram values
 * relevant for each thread into local registers to provide fast access.
 *
 * TODO: Reimplementing shared memory with proper reductions to avoid atomic
 * accesses might work faster in the end than using the local registers + atomic.
 *
 * Note: All arrays must be device-allocated
 *
 * @param hists array of input histograms
 * @param num_hists number of histograms
 * @param h_params histogram parameters
 * @param ce_vals output array of conditional entropy values
 */
__global__ void ConditionalEntropyKernel(const float *__restrict__ hists,
										 const size_t             num_hists,
										 const ConditionalEntropy h_params,
										 float *__restrict__ ce_vals)
{
	// Which histogram row this thread is summing
	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

	// Don't compute for out-of-bounds histograms
	if(idx / h_params.NumPhaseBins() >= num_hists)
	{
		return;
	}

	// Index in the histogram array corresponding to the start of this row
	const size_t offset = idx * h_params.NumMagBins();

	// This works if NumMagBins is fairly small.
	float  local_sum    = 0.0f;
	size_t num_mag_bins = h_params.NumMagBins();
	float  local_hists[128]; // assumes no more than 128 bins. Can't declare with num_mag_bins because c++ enforces known array sizes.
	for(size_t i = 0; i < num_mag_bins; i++)
	{
		local_hists[i] = hists[i + offset];
		local_sum += local_hists[i];
	}

	// Compute per-phase-bin conditional entropy
	// TODO: remove use of global mem?
	float p_j = local_sum; // Store p_j
	local_sum = 0.0f;
	for(size_t i = 0; i < num_mag_bins; i++)
	{
		float p_ij = local_hists[i];
		if(p_ij != 0)
		{
			local_sum += p_ij * logf(p_j / p_ij);
		}
	}

	// Accumulate per-phase-bin conditional entropy into total conditional
	// entropy for the histogram.
	// TODO: Replace with shared memory reduction of some kind. *yikes*
	size_t ce_idx = idx / h_params.NumPhaseBins();
	atomicAdd(&ce_vals[ce_idx], local_sum);
}

//
// Wrapper Functions
//

float *ConditionalEntropy::DeviceFoldAndBin(const float *times,
											const float *mags,
											const size_t length,
											const float *periods,
											const float *period_dts,
											const size_t num_periods,
											const size_t num_p_dts) const
{
	// Number of bytes of global memory required to store output
	size_t bytes = NumBins() * sizeof(float) * num_periods * num_p_dts;

	// Allocate and zero global memory for output histograms
	float *dev_hists;
	gpuErrchk(hipMalloc(&dev_hists, bytes));
	gpuErrchk(hipMemset(dev_hists, 0, bytes));

	// Number of threads and corresponding shared memory usage
	const size_t num_threads  = 512;
	const size_t shared_bytes = NumBins() * sizeof(uint32_t);

	// Grid to search over periods and time derivatives
	const dim3 grid_dim = dim3(num_periods, num_p_dts);

	// NOTE: A ConditionalEntropy object is small enough that we can pass it in
	//       the registers by dereferencing it.
	FoldBinKernel<<<grid_dim, num_threads, shared_bytes>>>(
		times, mags, length, periods, period_dts, *this, dev_hists);

	return dev_hists;
}

float *ConditionalEntropy::FoldAndBin(const float *times,
									  const float *mags,
									  const size_t length,
									  const float *periods,
									  const float *period_dts,
									  const size_t num_periods,
									  const size_t num_p_dts) const
{
	// Number of bytes of input data
	const size_t data_bytes = length * sizeof(float);

	// Allocate device pointers
	float *dev_times;
	float *dev_mags;
	float *dev_periods;
	float *dev_period_dts;
	gpuErrchk(hipMalloc(&dev_times, data_bytes));
	gpuErrchk(hipMalloc(&dev_mags, data_bytes));
	gpuErrchk(hipMalloc(&dev_periods, num_periods * sizeof(float)));
	gpuErrchk(hipMalloc(&dev_period_dts, num_p_dts * sizeof(float)));

	// Copy data to device memory
	gpuErrchk(hipMemcpy(dev_times, times, data_bytes, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_mags, mags, data_bytes, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_periods, periods, num_periods * sizeof(float),
						 hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_periods, period_dts, num_p_dts * sizeof(float),
						 hipMemcpyHostToDevice));

	float *dev_hists =
		DeviceFoldAndBin(dev_times, dev_mags, length, dev_periods,
						 dev_period_dts, num_periods, num_p_dts);

	// Allocate host histograms and copy from device
	size_t bytes = NumBins() * num_periods * num_p_dts * sizeof(float);
	float *hists = (float *) malloc(bytes);
	gpuErrchk(hipMemcpy(hists, dev_hists, bytes, hipMemcpyDeviceToHost));

	// Free GPU memory
	gpuErrchk(hipFree(dev_times));
	gpuErrchk(hipFree(dev_mags));
	gpuErrchk(hipFree(dev_periods));
	gpuErrchk(hipFree(dev_period_dts));
	gpuErrchk(hipFree(dev_hists));

	return hists;
}

float *ConditionalEntropy::DeviceCalcCEFromHists(const float *hists,
												 const size_t num_hists) const
{
	// Allocate global memory for output conditional entropy values
	float *dev_ces;
	gpuErrchk(hipMalloc(&dev_ces, num_hists * sizeof(float)));

	const size_t n_t = 512;
	const size_t n_b = ((num_hists * NumPhaseBins()) / n_t) + 1;

	// NOTE: A ConditionalEntropy object is small enough that we can pass it in
	//       the registers by dereferencing it.
	ConditionalEntropyKernel<<<n_b, n_t, n_t * sizeof(float)>>>(
		hists, num_hists, *this, dev_ces);

	return dev_ces;
}

float *ConditionalEntropy::CalcCEFromHists(const float *hists,
										   const size_t num_hists) const
{
	// Number of bytes in the histogram
	const size_t bytes = num_hists * NumBins() * sizeof(float);

	// Allocate device memory for histograms and copy over
	float *dev_hists;
	gpuErrchk(hipMalloc(&dev_hists, bytes));
	gpuErrchk(hipMemcpy(dev_hists, hists, bytes, hipMemcpyHostToDevice));

	float *dev_ces = DeviceCalcCEFromHists(dev_hists, num_hists);

	// Copy CEs to host
	float *ces = (float *) malloc(num_hists * sizeof(float));
	gpuErrchk(hipMemcpy(ces, dev_ces, num_hists * sizeof(float),
						 hipMemcpyDeviceToHost));

	// Free GPU memory
	gpuErrchk(hipFree(dev_hists));
	gpuErrchk(hipFree(dev_ces));

	return ces;
}

void ConditionalEntropy::CalcCEVals(float       *times,
									float       *mags,
									size_t       length,
									const float *periods,
									const float *period_dts,
									const size_t num_periods,
									const size_t num_p_dts,
									float       *ce_out) const
{
	CalcCEValsBatched(std::vector<float *>{times}, std::vector<float *>{mags},
					  std::vector<size_t>{length}, periods, period_dts,
					  num_periods, num_p_dts, ce_out);
}

float *ConditionalEntropy::CalcCEVals(float       *times,
									  float       *mags,
									  size_t       length,
									  const float *periods,
									  const float *period_dts,
									  const size_t num_periods,
									  const size_t num_p_dts) const
{
	return CalcCEValsBatched(std::vector<float *>{times},
							 std::vector<float *>{mags},
							 std::vector<size_t>{length}, periods, period_dts,
							 num_periods, num_p_dts);
}

void ConditionalEntropy::CalcCEValsBatched(const std::vector<float *> &times,
										   const std::vector<float *> &mags,
										   const std::vector<size_t>  &lengths,
										   const float *__restrict__ periods,
										   const float *__restrict__ period_dts,
										   const size_t num_periods,
										   const size_t num_p_dts,
										   float *__restrict__ ce_out) const
{
	// Size of one CE out array, and total CE output size.
	size_t ce_out_size   = num_periods * num_p_dts * sizeof(float);
	size_t ce_size_total = ce_out_size * lengths.size();

	// Copy trial information over
	float *dev_periods;
	float *dev_period_dts;
	gpuErrchk(hipMalloc(&dev_periods, num_periods * sizeof(float)));
	gpuErrchk(hipMalloc(&dev_period_dts, num_p_dts * sizeof(float)));
	gpuErrchk(hipMemcpy(dev_periods, periods, num_periods * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_period_dts, period_dts, num_p_dts * sizeof(float), hipMemcpyHostToDevice));

	// Intermediate histogram memory
	size_t num_hists  = num_periods * num_p_dts;
	size_t hist_bytes = NumBins() * sizeof(float) * num_hists;
	float *dev_hists;
	gpuErrchk(hipMalloc(&dev_hists, hist_bytes));

	// Intermediate conditional entropy memory
	float *dev_ces;
	gpuErrchk(hipMalloc(&dev_ces, ce_out_size));

	// Kernel launch information for the fold & bin step
	const size_t num_threads_fb  = 256;
	const size_t shared_bytes_fb = NumBins() * sizeof(uint32_t);
	const dim3   grid_dim_fb     = dim3(num_periods, num_p_dts);

	// Kernel launch information for the ce calculation step
	const size_t num_threads_ce = 256;
	const size_t num_blocks_ce  = ((num_hists * NumPhaseBins()) / num_threads_ce) + 1;

	// Buffer size (large enough for longest light curve)
	auto         max_length    = std::max_element(lengths.begin(), lengths.end());
	const size_t buffer_length = *max_length;
	const size_t buffer_bytes  = sizeof(float) * buffer_length;

	float *dev_times_buffer;
	float *dev_mags_buffer;
	gpuErrchk(hipMalloc(&dev_times_buffer, buffer_bytes));
	gpuErrchk(hipMalloc(&dev_mags_buffer, buffer_bytes));

	size_t total_elements = 0;
	for(size_t i = 0; i < lengths.size(); i++)
	{
		total_elements += lengths[i];
	}

	size_t contiguous_offset = 0;
	float *__restrict__ host_times_contiguous;
	float *__restrict__ host_mags_contiguous;
	gpuErrchk(hipHostAlloc((void **) &host_times_contiguous, total_elements * sizeof(float), hipHostMallocDefault));
	gpuErrchk(hipHostAlloc((void **) &host_mags_contiguous, total_elements * sizeof(float), hipHostMallocDefault));

	for(size_t i = 0; i < lengths.size(); i++)
	{
		memcpy(host_times_contiguous + contiguous_offset, times[i], lengths[i] * sizeof(float));
		memcpy(host_mags_contiguous + contiguous_offset, mags[i], lengths[i] * sizeof(float));
		contiguous_offset += lengths[i];
	}

	const size_t num_streams = 3;
	hipStream_t streams[num_streams];
	for(size_t i = 0; i < num_streams; i++)
	{
		gpuErrchk(hipStreamCreate(&streams[i]));
	}

	size_t curve_offset = 0;
	for(size_t batch_idx = 0; batch_idx < lengths.size(); batch_idx += num_streams)
	{
		for(size_t stream_idx = 0; stream_idx < num_streams; stream_idx++)
		{
			const size_t stream_batch_idx = batch_idx + stream_idx;

			if(stream_batch_idx >= lengths.size())
			{
				break;
			}

			const size_t curve_bytes = lengths[stream_batch_idx] * sizeof(float);
			gpuErrchk(hipMemcpyAsync(dev_times_buffer, host_times_contiguous + curve_offset, curve_bytes, hipMemcpyHostToDevice, streams[stream_idx]));
			gpuErrchk(hipMemcpyAsync(dev_mags_buffer, host_mags_contiguous + curve_offset, curve_bytes, hipMemcpyHostToDevice, streams[stream_idx]));

			gpuErrchk(hipMemsetAsync(dev_ces, 0, ce_out_size, streams[stream_idx]));

			FoldBinKernel<<<grid_dim_fb, num_threads_fb, shared_bytes_fb, streams[stream_idx]>>>(
				dev_times_buffer, dev_mags_buffer, lengths[stream_batch_idx], dev_periods,
				dev_period_dts, *this, dev_hists);

			ConditionalEntropyKernel<<<num_blocks_ce, num_threads_ce, 0, streams[stream_idx]>>>(
				dev_hists, num_hists, *this, dev_ces);

			gpuErrchk(hipMemcpyAsync(&ce_out[stream_batch_idx * num_hists], dev_ces, ce_out_size, hipMemcpyDeviceToHost, streams[stream_idx]));

			curve_offset += curve_bytes / sizeof(float);
		}
	}

	for(size_t i = 0; i < num_streams; ++i)
	{
		gpuErrchk(hipStreamSynchronize(streams[i]));
		gpuErrchk(hipStreamDestroy(streams[i]));
	}

	gpuErrchk(hipFree(dev_periods));
	gpuErrchk(hipFree(dev_period_dts));
	gpuErrchk(hipFree(dev_hists));
	gpuErrchk(hipFree(dev_ces));
	gpuErrchk(hipFree(dev_times_buffer));
	gpuErrchk(hipFree(dev_mags_buffer));
	hipHostFree(host_times_contiguous);
	hipHostFree(host_mags_contiguous);
}


float *ConditionalEntropy::CalcCEValsBatched(const std::vector<float *> &times,
											 const std::vector<float *> &mags,
											 const std::vector<size_t>  &lengths,
											 const float                *periods,
											 const float                *period_dts,
											 const size_t                num_periods,
											 const size_t                num_p_dts) const
{
	// Size of one CE out array, and total CE output size.
	size_t ce_out_size   = num_periods * num_p_dts * sizeof(float);
	size_t ce_size_total = ce_out_size * lengths.size();

	// Allocate host memory for output CE values.
	float *ce_out = (float *) malloc(ce_size_total);

	// Perform CE calculation.
	CalcCEValsBatched(times, mags, lengths, periods, period_dts, num_periods,
					  num_p_dts, ce_out);

	return ce_out;
}
