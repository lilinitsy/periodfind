#include "hip/hip_runtime.h"
// Copyright 2020 California Institute of Technology. All rights reserved.
// Use of this source code is governed by a BSD-style license that can be
// found in the LICENSE file.
// Author: Ethan Jaszewski

#include "ls.h"

#include <algorithm>

#include <cstdio>

#include "hip/hip_runtime.h"
#include "math.h"

#include "errchk.cuh"

const float TWO_PI = M_PI * 2.0;

//
// Simple LombScargle Function Definitions
//

LombScargle::LombScargle() {}

//
// CUDA Kernels
//

__global__ void LombScargleKernel(const float* __restrict__ times,
                                  const float* __restrict__ mags,
                                  const size_t length,
                                  const float* __restrict__ periods,
                                  const float* __restrict__ period_dts,
                                  const size_t num_periods,
                                  const size_t num_period_dts,
                                  const LombScargle params,
                                  float* periodogram) {
    const size_t thread_x = threadIdx.x + blockIdx.x * blockDim.x;
    const size_t thread_y = threadIdx.y + blockIdx.y * blockDim.y;

    if (thread_x >= num_periods || thread_y >= num_period_dts) {
        return;
    }

    // Period and period time derivative
    const float period = periods[thread_x];
    const float period_dt = period_dts[thread_y];

    // Time derivative correction factor.
    const float pdt_corr = (period_dt / period) * 0.5f;

    float mag_cos = 0.0f;
    float mag_sin = 0.0f;
    float cos_cos = 0.0f;
    float cos_sin = 0.0f;

    float cos, sin, i_part;

#pragma unroll
    for (size_t idx = 0; idx < length; idx++) {
        float t = times[idx];
        float mag = mags[idx];

        float t_corr = t - pdt_corr * t * t;
        float folded = fabsf(modff(t_corr / period, &i_part));

        __sincosf(TWO_PI * folded, &sin, &cos);

        mag_cos += mag * cos;
        mag_sin += mag * sin;
        cos_cos += cos * cos;
        cos_sin += cos * sin;
    }

    float sin_sin = static_cast<float>(length) - cos_cos;

    float cos_tau, sin_tau;
    __sincosf(0.5f * atan2f(2.0f * cos_sin, cos_cos - sin_sin), &sin_tau,
              &cos_tau);

    float numerator_l = cos_tau * mag_cos + sin_tau * mag_sin;
    numerator_l *= numerator_l;

    float numerator_r = cos_tau * mag_sin - sin_tau * mag_cos;
    numerator_r *= numerator_r;

    float denominator_l = cos_tau * cos_tau * cos_cos
                          + 2 * cos_tau * sin_tau * cos_sin
                          + sin_tau * sin_tau * sin_sin;

    float denominator_r = cos_tau * cos_tau * sin_sin
                          - 2 * cos_tau * sin_tau * cos_sin
                          + sin_tau * sin_tau * cos_cos;

    periodogram[thread_x * num_period_dts + thread_y] =
        0.5 * ((numerator_l / denominator_l) + (numerator_r / denominator_r));
}

//
// Wrapper Functions
//

float* LombScargle::DeviceCalcLS(const float* times,
                                 const float* mags,
                                 const size_t length,
                                 const float* periods,
                                 const float* period_dts,
                                 const size_t num_periods,
                                 const size_t num_p_dts) const {
    float* periodogram;
    gpuErrchk(
        hipMalloc(&periodogram, num_periods * num_p_dts * sizeof(float)));

    const size_t x_threads = 256;
    const size_t y_threads = 1;
    const size_t x_blocks = ((num_periods + x_threads - 1) / x_threads);
    const size_t y_blocks = ((num_p_dts + y_threads - 1) / y_threads);

    const dim3 block_dim = dim3(x_threads, y_threads);
    const dim3 grid_dim = dim3(x_blocks, y_blocks);

    LombScargleKernel<<<grid_dim, block_dim>>>(times, mags, length, periods,
                                               period_dts, num_periods,
                                               num_p_dts, *this, periodogram);

    return periodogram;
}

void LombScargle::CalcLS(float* times,
                         float* mags,
                         size_t length,
                         const float* periods,
                         const float* period_dts,
                         const size_t num_periods,
                         const size_t num_p_dts,
                         float* per_out) const {
    CalcLSBatched(std::vector<float*>{times}, std::vector<float*>{mags},
                  std::vector<size_t>{length}, periods, period_dts, num_periods,
                  num_p_dts, per_out);
}

float* LombScargle::CalcLS(float* times,
                           float* mags,
                           size_t length,
                           const float* periods,
                           const float* period_dts,
                           const size_t num_periods,
                           const size_t num_p_dts) const {
    return CalcLSBatched(std::vector<float*>{times}, std::vector<float*>{mags},
                         std::vector<size_t>{length}, periods, period_dts,
                         num_periods, num_p_dts);
}

void LombScargle::CalcLSBatched(const std::vector<float*>& times,
                                const std::vector<float*>& mags,
                                const std::vector<size_t>& lengths,
                                const float* periods,
                                const float* period_dts,
                                const size_t num_periods,
                                const size_t num_p_dts,
                                float* per_out) const {
    // TODO: Look at ways of batching data transfer.

    // Size of one periodogram out array, and total periodogram output size.
    const size_t per_points = num_periods * num_p_dts;
    const size_t per_out_size = per_points * sizeof(float);
    const size_t per_points_doubled = 2 * per_points;
    const size_t num_streams = 3;

    // Buffer size (large enough for longest light curve)
    auto max_length = std::max_element(lengths.begin(), lengths.end());
    const size_t buffer_length = *max_length;
    const size_t buffer_bytes = sizeof(float) * buffer_length;
    const size_t buffer_length_doubled = 2 * buffer_length;

    // Trial information
    float* dev_periods;
    float* dev_period_dts;

    // Intermediate conditional entropy memory
    float* dev_per;

    // Device data
    float* dev_times_buffer;
    float* dev_mags_buffer;

    // Kernel launch information
    const size_t x_threads = 512;
    const size_t y_threads = 1;
    const size_t x_blocks = ((num_periods + x_threads - 1) / x_threads);
    const size_t y_blocks = ((num_p_dts + y_threads - 1) / y_threads);
    const dim3 block_dim = dim3(x_threads, y_threads);
    const dim3 grid_dim = dim3(x_blocks, y_blocks);

    // Create 3 cuda streams to pipeline async operations; a loop would be
    // cleaner, but slowed execution
    hipStream_t stream1;
    hipStream_t stream2;
    hipStream_t stream3;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);

    // Perform all allocations
    gpuErrchk(hipMalloc(&dev_periods, num_periods * sizeof(float)));
    gpuErrchk(hipMalloc(&dev_period_dts, num_p_dts * sizeof(float)));
    gpuErrchk(hipMemcpy(dev_periods, periods, num_periods * sizeof(float),
                         hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dev_period_dts, period_dts, num_p_dts * sizeof(float),
                         hipMemcpyHostToDevice));
    gpuErrchk(hipMalloc(&dev_per, per_out_size * num_streams));
    gpuErrchk(hipMalloc(&dev_times_buffer, buffer_bytes * num_streams));
    gpuErrchk(hipMalloc(&dev_mags_buffer, buffer_bytes * num_streams));

    /*
    Zero conditional entropy output
    This strictly does not seem to be necessary since every entry in dev_per
    should be written to, but it's good to do at initialization to avoid
    garbage being accidentally set. It can be done per loop too, but it
    probably isn't necessary and introduces overhead */
    gpuErrchk(hipMemset(dev_per, 0, per_out_size * num_streams));

    // Changing the code to use an inner loop increases runtime quite a bit
    // It might be that CUDA's async calls aren't optimized by the compiler when
    // doing that
#pragma unroll
    for (size_t i = 0; i < lengths.size(); i += num_streams) {
        // Copy light curve into device buffer
        const size_t i_plus_1 = i + 1;
        const size_t i_plus_2 = i + 2;
        const size_t curve_bytes_i = lengths[i] * sizeof(float);

        hipMemcpyAsync(dev_times_buffer, times[i], curve_bytes_i,
                        hipMemcpyHostToDevice, stream1);
        hipMemcpyAsync(dev_mags_buffer, mags[i], curve_bytes_i,
                        hipMemcpyHostToDevice, stream1);

        LombScargleKernel<<<grid_dim, block_dim, 0, stream1>>>(
            dev_times_buffer, dev_mags_buffer, lengths[i], dev_periods,
            dev_period_dts, num_periods, num_p_dts, *this, dev_per);

        if (i_plus_1 < lengths.size()) {
            const size_t curve_bytes_next = lengths[i_plus_1] * sizeof(float);
            gpuErrchk(hipMemcpyAsync(dev_times_buffer + buffer_length,
                                      times[i_plus_1], curve_bytes_next,
                                      hipMemcpyHostToDevice, stream2));
            gpuErrchk(hipMemcpyAsync(dev_mags_buffer + buffer_length,
                                      mags[i_plus_1], curve_bytes_next,
                                      hipMemcpyHostToDevice, stream2));

            LombScargleKernel<<<grid_dim, block_dim, 0, stream2>>>(
                dev_times_buffer + buffer_length,
                dev_mags_buffer + buffer_length, lengths[i_plus_1], dev_periods,
                dev_period_dts, num_periods, num_p_dts, *this,
                dev_per + per_points);
        }

        if (i_plus_2 < lengths.size()) {
            const size_t curve_bytes_next = lengths[i_plus_2] * sizeof(float);
            gpuErrchk(hipMemcpyAsync(dev_times_buffer + buffer_length_doubled,
                                      times[i_plus_2], curve_bytes_next,
                                      hipMemcpyHostToDevice, stream3));
            gpuErrchk(hipMemcpyAsync(dev_mags_buffer + buffer_length_doubled,
                                      mags[i_plus_2], curve_bytes_next,
                                      hipMemcpyHostToDevice, stream3));

            LombScargleKernel<<<grid_dim, block_dim, 0, stream3>>>(
                dev_times_buffer + buffer_length_doubled,
                dev_mags_buffer + buffer_length_doubled, lengths[i_plus_2],
                dev_periods, dev_period_dts, num_periods, num_p_dts, *this,
                dev_per + per_points_doubled);
        }

        // Copy periodogram back to host
        gpuErrchk(hipMemcpyAsync(&per_out[i * per_points], dev_per,
                                  per_out_size, hipMemcpyDeviceToHost,
                                  stream1));

        if (i_plus_1 < lengths.size()) {
            gpuErrchk(hipMemcpyAsync(&per_out[i_plus_1 * per_points],
                                      dev_per + per_points, per_out_size,
                                      hipMemcpyDeviceToHost, stream2));
        }

        if (i_plus_2 < lengths.size()) {
            gpuErrchk(hipMemcpyAsync(
                &per_out[i_plus_2 * per_points], dev_per + per_points_doubled,
                per_out_size, hipMemcpyDeviceToHost, stream3));
        }
    }

    gpuErrchk(hipStreamSynchronize(stream1));
    gpuErrchk(hipStreamSynchronize(stream2));
    gpuErrchk(hipStreamSynchronize(stream3));
    gpuErrchk(hipStreamDestroy(stream1));
    gpuErrchk(hipStreamDestroy(stream2));
    gpuErrchk(hipStreamDestroy(stream3));

    // Free all of the GPU memory
    gpuErrchk(hipFree(dev_periods));
    gpuErrchk(hipFree(dev_period_dts));
    gpuErrchk(hipFree(dev_per));
    gpuErrchk(hipFree(dev_times_buffer));
    gpuErrchk(hipFree(dev_mags_buffer));
}

float* LombScargle::CalcLSBatched(const std::vector<float*>& times,
                                  const std::vector<float*>& mags,
                                  const std::vector<size_t>& lengths,
                                  const float* periods,
                                  const float* period_dts,
                                  const size_t num_periods,
                                  const size_t num_p_dts) const {
    // Size of one periodogram out array, and total periodogram output size.
    size_t per_points = num_periods * num_p_dts;
    size_t per_out_size = per_points * sizeof(float);
    size_t per_size_total = per_out_size * lengths.size();

    // Allocate the output CE array so we can copy to it.
    float* per_out = (float*)malloc(per_size_total);

    CalcLSBatched(times, mags, lengths, periods, period_dts, num_periods,
                  num_p_dts, per_out);

    return per_out;
}