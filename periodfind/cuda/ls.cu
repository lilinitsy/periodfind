#include "hip/hip_runtime.h"
// Copyright 2020 California Institute of Technology. All rights reserved.
// Use of this source code is governed by a BSD-style license that can be
// found in the LICENSE file.
// Author: Ethan Jaszewski

#include "ls.h"

#include <algorithm>

#include <chrono>
#include <cstdio>

#include "hip/hip_runtime.h"
#include "math.h"

#include "errchk.cuh"

const float TWO_PI = M_PI * 2.0;

//
// Simple LombScargle Function Definitions
//

LombScargle::LombScargle()
{
}

//
// CUDA Kernels
//

__global__ void LombScargleKernel(const float *times,
								  const float *mags,
								  const size_t length,
								  const float *periods,
								  const float *period_dts,
								  const size_t num_periods,
								  const size_t num_period_dts,
								  float       *periodogram)
{
	const size_t thread_x = threadIdx.x + blockIdx.x * blockDim.x;
	const size_t thread_y = threadIdx.y + blockIdx.y * blockDim.y;

	if(thread_x >= num_periods || thread_y >= num_period_dts)
	{
		return;
	}

	// Period and period time derivative
	const float period    = periods[thread_x];
	const float period_dt = period_dts[thread_y];

	// Time derivative correction factor.
	const float pdt_corr = (period_dt / period) / 2;

	float mag_cos = 0.0;
	float mag_sin = 0.0;
	float cos_cos = 0.0;
	float cos_sin = 0.0;

	float cos, sin, i_part;

	for(size_t idx = 0; idx < length; idx++)
	{
		float t   = times[idx];
		float mag = mags[idx];

		float t_corr = t - pdt_corr * t * t;
		float folded = fabsf(modff(t_corr / period, &i_part));

		sincosf(TWO_PI * folded, &sin, &cos);

		mag_cos += mag * cos;
		mag_sin += mag * sin;
		cos_cos += cos * cos;
		cos_sin += cos * sin;
	}

	float sin_sin = static_cast<float>(length) - cos_cos;

	float cos_tau, sin_tau;
	sincosf(0.5 * atan2f(2.0 * cos_sin, cos_cos - sin_sin), &sin_tau, &cos_tau);

	float numerator_l = cos_tau * mag_cos + sin_tau * mag_sin;
	numerator_l *= numerator_l;

	float numerator_r = cos_tau * mag_sin - sin_tau * mag_cos;
	numerator_r *= numerator_r;

	float denominator_l = cos_tau * cos_tau * cos_cos + 2 * cos_tau * sin_tau * cos_sin + sin_tau * sin_tau * sin_sin;

	float denominator_r = cos_tau * cos_tau * sin_sin - 2 * cos_tau * sin_tau * cos_sin + sin_tau * sin_tau * cos_cos;

	periodogram[thread_x * num_period_dts + thread_y] =
		0.5 * ((numerator_l / denominator_l) + (numerator_r / denominator_r));
}

__global__ void LombScargleKernelBatched(const float  *times,
										 const float  *mags,
										 const size_t *lengths,
										 const float  *periods,
										 const float  *period_dts,
										 const size_t  num_periods,
										 const size_t  num_period_dts,
										 const size_t  num_curves,
										 float        *periodogram)
{
	const size_t thread_x = threadIdx.x + blockIdx.x * blockDim.x;
	const size_t thread_y = threadIdx.y + blockIdx.y * blockDim.y;

    #pragma unroll
	for(size_t curve_idx = 0; curve_idx < num_curves; curve_idx++)
	{
		if(thread_x >= num_periods || thread_y >= num_period_dts)
		{
			return;
		}

		const size_t length = lengths[curve_idx];

		size_t offset = 0;
		for(size_t i = 0; i < curve_idx; i++)
		{
			offset += lengths[i];
		}

		// Period and period time derivative
		const float period    = periods[thread_x];
		const float period_dt = period_dts[thread_y];

		// Time derivative correction factor.
		const float pdt_corr = (period_dt / period) / 2;

		float mag_cos = 0.0;
		float mag_sin = 0.0;
		float cos_cos = 0.0;
		float cos_sin = 0.0;

		float cos, sin, i_part;

		for(size_t idx = 0; idx < length; idx++)
		{
			float t   = times[idx + offset];
			float mag = mags[idx + offset];

			float t_corr = t - pdt_corr * t * t;
			float folded = fabsf(modff(t_corr / period, &i_part));

			__sincosf(TWO_PI * folded, &sin, &cos);

			mag_cos += mag * cos;
			mag_sin += mag * sin;
			cos_cos += cos * cos;
			cos_sin += cos * sin;
		}

		float sin_sin = static_cast<float>(length) - cos_cos;

		float cos_tau, sin_tau;
		__sincosf(0.5 * atan2f(2.0 * cos_sin, cos_cos - sin_sin), &sin_tau, &cos_tau);

		float numerator_l = cos_tau * mag_cos + sin_tau * mag_sin;
		float numerator_r = cos_tau * mag_sin - sin_tau * mag_cos;
		numerator_l *= numerator_l;
		numerator_r *= numerator_r;

		float denominator_l = cos_tau * cos_tau * cos_cos + 2 * cos_tau * sin_tau * cos_sin + sin_tau * sin_tau * sin_sin;
		float denominator_r = cos_tau * cos_tau * sin_sin - 2 * cos_tau * sin_tau * cos_sin + sin_tau * sin_tau * cos_cos;

		periodogram[(curve_idx * num_periods * num_period_dts) + (thread_x * num_period_dts + thread_y)] =
			0.5 * ((numerator_l / denominator_l) + (numerator_r / denominator_r));
	}
}

//
// Wrapper Functions
//

float *LombScargle::DeviceCalcLS(const float *times,
								 const float *mags,
								 const size_t length,
								 const float *periods,
								 const float *period_dts,
								 const size_t num_periods,
								 const size_t num_p_dts) const
{
	float *periodogram;
	gpuErrchk(
		hipMalloc(&periodogram, num_periods * num_p_dts * sizeof(float)));

	const size_t x_threads = 256;
	const size_t y_threads = 1;
	const size_t x_blocks  = ((num_periods + x_threads - 1) / x_threads);
	const size_t y_blocks  = ((num_p_dts + y_threads - 1) / y_threads);

	const dim3 block_dim = dim3(x_threads, y_threads);
	const dim3 grid_dim  = dim3(x_blocks, y_blocks);

	LombScargleKernel<<<grid_dim, block_dim>>>(times, mags, length, periods,
											   period_dts, num_periods,
											   num_p_dts, periodogram);

	return periodogram;
}

void LombScargle::CalcLS(float       *times,
						 float       *mags,
						 size_t       length,
						 const float *periods,
						 const float *period_dts,
						 const size_t num_periods,
						 const size_t num_p_dts,
						 float       *per_out) const
{
	CalcLSBatched(std::vector<float *>{times}, std::vector<float *>{mags},
				  std::vector<size_t>{length}, periods, period_dts, num_periods,
				  num_p_dts, per_out);
}

float *LombScargle::CalcLS(float       *times,
						   float       *mags,
						   size_t       length,
						   const float *periods,
						   const float *period_dts,
						   const size_t num_periods,
						   const size_t num_p_dts) const
{
	return CalcLSBatched(std::vector<float *>{times}, std::vector<float *>{mags},
						 std::vector<size_t>{length}, periods, period_dts,
						 num_periods, num_p_dts);
}

// There's no real point in having times and mags be a vector
// because the lengths vector already keeps a list of how long each
// float* is. They should be reorganized to be a contiguous block, but that data
// processing needs to happen on the python side. On the other hand, this takes
// less than a tenth of a second to convert
void LombScargle::CalcLSBatched(const std::vector<float *> &times,
								const std::vector<float *> &mags,
								const std::vector<size_t>  &lengths,
								const float                *periods,
								const float                *period_dts,
								const size_t                num_periods,
								const size_t                num_p_dts,
								float                      *per_out) const
{
	const size_t num_curves     = 8;
	size_t       per_points     = num_periods * num_p_dts;
	size_t       per_out_size   = num_curves * per_points * sizeof(float);
	size_t       per_size_total = per_points * sizeof(float) * lengths.size();

	// Allocate device memory
	float *dev_periods;
	float *dev_period_dts;
	gpuErrchk(hipMalloc(&dev_periods, num_periods * sizeof(float)));
	gpuErrchk(hipMalloc(&dev_period_dts, num_p_dts * sizeof(float)));
	gpuErrchk(hipMemcpy(dev_periods, periods, num_periods * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_period_dts, period_dts, num_p_dts * sizeof(float), hipMemcpyHostToDevice));

	float *dev_per;
	gpuErrchk(hipMalloc(&dev_per, per_out_size));

	const size_t x_threads = 1024;
	const size_t y_threads = 1;
	const size_t x_blocks  = (num_periods + x_threads - 1) / x_threads;
	const size_t y_blocks  = (num_p_dts + y_threads - 1) / y_threads;
	const dim3   block_dim = dim3(x_threads, y_threads);
	const dim3   grid_dim  = dim3(x_blocks, y_blocks);

	// Determine the maximum buffer size needed
	auto         max_length    = std::max_element(lengths.begin(), lengths.end());
	const size_t buffer_length = *max_length;
	const size_t buffer_bytes  = num_curves * buffer_length * sizeof(float);

	// Allocate device buffers
	float  *dev_times_buffer;
	float  *dev_mags_buffer;
	size_t *dev_lengths_buffer;
	gpuErrchk(hipMalloc(&dev_times_buffer, buffer_bytes));
	gpuErrchk(hipMalloc(&dev_mags_buffer, buffer_bytes));
	gpuErrchk(hipMalloc(&dev_lengths_buffer, num_curves * sizeof(size_t)));

	std::chrono::high_resolution_clock::time_point start =
		std::chrono::high_resolution_clock::now();

	// Calculate the total number of elements for contiguous arrays
	size_t total_elements = 0;
	for(size_t i = 0; i < lengths.size(); i++)
	{
		total_elements += lengths[i];
	}

	// Allocate and copy data to contiguous memory on host
	float *host_times_contiguous = new float[total_elements];
	float *host_mags_contiguous  = new float[total_elements];
	size_t contiguous_offset     = 0;

	for(size_t i = 0; i < lengths.size(); i++)
	{
		memcpy(host_times_contiguous + contiguous_offset, times[i], lengths[i] * sizeof(float));
		memcpy(host_mags_contiguous + contiguous_offset, mags[i], lengths[i] * sizeof(float));
		contiguous_offset += lengths[i];
	}

	printf("lengths.size: %zu\n", lengths.size());

	printf("num_curves: %lu\n", num_curves);
	printf("buffer length (max_length): %lu\n", buffer_length);
	printf("buffer_bytes: %lu\n", buffer_bytes);
	printf("lengths.size(), bytes: %lu\t%lu\n", lengths.size(), lengths.size() * sizeof(size_t));
	printf("times.size(), bytes: %lu\t%lu\n", times.size(), times.size() * sizeof(float *));
	printf("mags.size(), bytes: %lu\t%lu\n", mags.size(), mags.size() * sizeof(float *));
	printf("per_out_size: %lu\n", per_out_size);
	printf("per_size_total: %lu bytes = %lu Kb = %lu Mb\n", per_size_total, per_size_total / 1024, per_size_total / (1024 * 1024));
	printf("num_periods: %lu\n", num_periods);
	printf("num_p_dts: %lu\n", num_p_dts);
	printf("per_points: %lu\n", per_points);

	std::chrono::high_resolution_clock::time_point end     = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double>                  elapsed = std::chrono::duration_cast<std::chrono::duration<double>>(end - start);
	printf("Time for extra cpu-side memcpy (to be removed): %f seconds\n", elapsed.count());

	size_t curve_offset = 0;
	for(size_t batch_idx = 0; batch_idx < lengths.size(); batch_idx += num_curves)
	{
		size_t curve_bytes = 0;

		for(size_t i = 0; i < num_curves && batch_idx + i < lengths.size(); i++)
		{
			curve_bytes += lengths[batch_idx + i] * sizeof(float);
		}

		hipMemcpy(dev_times_buffer, host_times_contiguous + curve_offset, curve_bytes, hipMemcpyHostToDevice);
		hipMemcpy(dev_mags_buffer, host_mags_contiguous + curve_offset, curve_bytes, hipMemcpyHostToDevice);
		hipMemcpy(dev_lengths_buffer, &lengths[batch_idx], num_curves * sizeof(size_t), hipMemcpyHostToDevice);

		LombScargleKernelBatched<<<grid_dim, block_dim>>>(
			dev_times_buffer,
			dev_mags_buffer,
			dev_lengths_buffer,
			dev_periods,
			dev_period_dts,
			num_periods,
			num_p_dts,
			num_curves,
			dev_per);

		hipMemcpy(&per_out[batch_idx * per_points], dev_per, per_out_size, hipMemcpyDeviceToHost);

		curve_offset += curve_bytes / sizeof(float);
	}

	// Free host and device memory
	delete[] host_times_contiguous;
	delete[] host_mags_contiguous;
	gpuErrchk(hipFree(dev_periods));
	gpuErrchk(hipFree(dev_period_dts));
	gpuErrchk(hipFree(dev_per));
	gpuErrchk(hipFree(dev_lengths_buffer));
	gpuErrchk(hipFree(dev_times_buffer));
	gpuErrchk(hipFree(dev_mags_buffer));
}

float *LombScargle::CalcLSBatched(const std::vector<float *> &times,
								  const std::vector<float *> &mags,
								  const std::vector<size_t>  &lengths,
								  const float                *periods,
								  const float                *period_dts,
								  const size_t                num_periods,
								  const size_t                num_p_dts) const
{
	// Size of one periodogram out array, and total periodogram output size.
	size_t per_points     = num_periods * num_p_dts;
	size_t per_out_size   = per_points * sizeof(float);
	size_t per_size_total = per_out_size * lengths.size();

	// Allocate the output CE array so we can copy to it.
	float *per_out = (float *) malloc(per_size_total);

	CalcLSBatched(times, mags, lengths, periods, period_dts, num_periods,
				  num_p_dts, per_out);

	return per_out;
}