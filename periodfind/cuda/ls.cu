#include "hip/hip_runtime.h"
// Copyright 2020 California Institute of Technology. All rights reserved.
// Use of this source code is governed by a BSD-style license that can be
// found in the LICENSE file.
// Author: Ethan Jaszewski

#include "ls.h"

#include <algorithm>

#include <cstdio>

#include "hip/hip_runtime.h"
#include "math.h"

#include "errchk.cuh"

const float TWO_PI = M_PI * 2.0;

//
// Simple LombScargle Function Definitions
//

LombScargle::LombScargle() {}

//
// CUDA Kernels
//

__global__ void LombScargleKernel(const float* times,
                                  const float* mags,
                                  const size_t length,
                                  const float* periods,
                                  const float* period_dts,
                                  const size_t num_periods,
                                  const size_t num_period_dts,
                                  const LombScargle params,
                                  float* periodogram) {
    const size_t thread_x = threadIdx.x + blockIdx.x * blockDim.x;
    const size_t thread_y = threadIdx.y + blockIdx.y * blockDim.y;

    if (thread_x >= num_periods || thread_y >= num_period_dts) {
        return;
    }

    // Period and period time derivative
    const float period = periods[thread_x];
    const float period_dt = period_dts[thread_y];

    // Time derivative correction factor.
    const float pdt_corr = (period_dt / period) / 2;

    float mag_cos = 0.0;
    float mag_sin = 0.0;
    float cos_cos = 0.0;
    float cos_sin = 0.0;

    float cos, sin, i_part;

    for (size_t idx = 0; idx < length; idx++) {
        float t = times[idx];
        float mag = mags[idx];

        float t_corr = t - pdt_corr * t * t;
        float folded = fabsf(modff(t_corr / period, &i_part));

        sincosf(TWO_PI * folded, &sin, &cos);

        mag_cos += mag * cos;
        mag_sin += mag * sin;
        cos_cos += cos * cos;
        cos_sin += cos * sin;
    }

    float sin_sin = static_cast<float>(length) - cos_cos;

    float cos_tau, sin_tau;
    sincosf(0.5 * atan2f(2.0 * cos_sin, cos_cos - sin_sin), &sin_tau, &cos_tau);

    float numerator_l = cos_tau * mag_cos + sin_tau * mag_sin;
    numerator_l *= numerator_l;

    float numerator_r = cos_tau * mag_sin - sin_tau * mag_cos;
    numerator_r *= numerator_r;

    float denominator_l = cos_tau * cos_tau * cos_cos
                          + 2 * cos_tau * sin_tau * cos_sin
                          + sin_tau * sin_tau * sin_sin;

    float denominator_r = cos_tau * cos_tau * sin_sin
                          - 2 * cos_tau * sin_tau * cos_sin
                          + sin_tau * sin_tau * cos_cos;

    periodogram[thread_x * num_period_dts + thread_y] =
        0.5 * ((numerator_l / denominator_l) + (numerator_r / denominator_r));
}

__global__ void LombScargleKernelBatched(const float* times,
                                         const float* mags,
                                         const size_t* lengths,
                                         const float* periods,
                                         const float* period_dts,
                                         const size_t num_periods,
                                         const size_t num_period_dts,
                                         const size_t num_curves,
                                         const size_t max_length,
                                         const LombScargle params,
                                         float* periodogram) {
    const size_t thread_x = threadIdx.x + blockIdx.x * blockDim.x;
    const size_t thread_y = threadIdx.y + blockIdx.y * blockDim.y;

    if (thread_x >= num_periods || thread_y >= num_period_dts) {
        return;
    }

    //printf("KERNEL: max_length %lu, num_curves %lu, thread_x %lu, thread_y %lu\n", max_length, num_curves, thread_x, thread_y);

    for (size_t curve_idx = 0; curve_idx < num_curves; ++curve_idx) {
        size_t offset = 0;
        for (size_t i = 0; i < curve_idx; ++i) {
            offset += lengths[i];
        }

        //printf("KERNEL: offset %lu, curve_idx %lu\n", offset, curve_idx);

        if (curve_idx >= num_curves) {
            //printf("Illegal access: curve_idx %lu exceeds num_curves %lu\n", curve_idx, num_curves);
            return;
        }

        const float* times_cur = times + offset;
        const float* mags_cur = mags + offset;

        const size_t length = lengths[curve_idx];
        //printf("KERNEL: length %lu\n", length);

        if (length > max_length) {
            //printf("Illegal memory access: length %lu exceeds max_length %lu\n", length, max_length);
            return;
        }

        // Period and period time derivative
        const float period = periods[thread_x];
        const float period_dt = period_dts[thread_y];

        // Time derivative correction factor.
        const float pdt_corr = (period_dt / period) / 2;

        float mag_cos = 0.0, mag_sin = 0.0, cos_cos = 0.0, cos_sin = 0.0;
        float cos, sin, i_part;

        for (size_t idx = 0; idx < length; idx++) {
            if (idx >= max_length) {
                //printf("Illegal memory access: idx %lu exceeds max_length %lu\n", idx, max_length);
                return;
            }

            float t = times_cur[idx];
            float mag = mags_cur[idx];

            float t_corr = t - pdt_corr * t * t;
            float folded = fabsf(modff(t_corr / period, &i_part));

            sincosf(TWO_PI * folded, &sin, &cos);

            mag_cos += mag * cos;
            mag_sin += mag * sin;
            cos_cos += cos * cos;
            cos_sin += cos * sin;
        }

        float sin_sin = static_cast<float>(length) - cos_cos;
        float cos_tau, sin_tau;
        sincosf(0.5 * atan2f(2.0 * cos_sin, cos_cos - sin_sin), &sin_tau, &cos_tau);

        float numerator_l = cos_tau * mag_cos + sin_tau * mag_sin;
        numerator_l *= numerator_l;

        float numerator_r = cos_tau * mag_sin - sin_tau * mag_cos;
        numerator_r *= numerator_r;

        float denominator_l = cos_tau * cos_cos + sin_tau * sin_sin + 2 * cos_tau * sin_tau * cos_sin;

        float denominator_r = cos_tau * sin_sin - sin_tau * cos_cos - 2 * cos_tau * sin_tau * cos_sin;

        periodogram[curve_idx * num_periods * num_period_dts + thread_x * num_period_dts + thread_y] =
            0.5 * ((numerator_l / denominator_l) + (numerator_r / denominator_r));
    }
}





//
// Wrapper Functions
//

float* LombScargle::DeviceCalcLS(const float* times,
                                 const float* mags,
                                 const size_t length,
                                 const float* periods,
                                 const float* period_dts,
                                 const size_t num_periods,
                                 const size_t num_p_dts) const {
    float* periodogram;
    gpuErrchk(
        hipMalloc(&periodogram, num_periods * num_p_dts * sizeof(float)));

    const size_t x_threads = 256;
    const size_t y_threads = 1;
    const size_t x_blocks = ((num_periods + x_threads - 1) / x_threads);
    const size_t y_blocks = ((num_p_dts + y_threads - 1) / y_threads);

    const dim3 block_dim = dim3(x_threads, y_threads);
    const dim3 grid_dim = dim3(x_blocks, y_blocks);

    LombScargleKernel<<<grid_dim, block_dim>>>(times, mags, length, periods,
                                               period_dts, num_periods,
                                               num_p_dts, *this, periodogram);

    return periodogram;
}

void LombScargle::CalcLS(float* times,
                         float* mags,
                         size_t length,
                         const float* periods,
                         const float* period_dts,
                         const size_t num_periods,
                         const size_t num_p_dts,
                         float* per_out) const {
    CalcLSBatched(std::vector<float*>{times}, std::vector<float*>{mags},
                  std::vector<size_t>{length}, periods, period_dts, num_periods,
                  num_p_dts, per_out);
}

float* LombScargle::CalcLS(float* times,
                           float* mags,
                           size_t length,
                           const float* periods,
                           const float* period_dts,
                           const size_t num_periods,
                           const size_t num_p_dts) const {
    return CalcLSBatched(std::vector<float*>{times}, std::vector<float*>{mags},
                         std::vector<size_t>{length}, periods, period_dts,
                         num_periods, num_p_dts);
}

void LombScargle::CalcLSBatched(const std::vector<float*>& times,
                                const std::vector<float*>& mags,
                                const std::vector<size_t>& lengths,
                                const float* periods,
                                const float* period_dts,
                                const size_t num_periods,
                                const size_t num_p_dts,
                                float* per_out) const {
    // Size of one periodogram out array, and total periodogram output size.
    size_t per_points = num_periods * num_p_dts;
    size_t per_out_size = per_points * sizeof(float);
    const size_t num_curves = 1;

    // Copy trial information over
    float* dev_periods;
    float* dev_period_dts;
    gpuErrchk(hipMalloc(&dev_periods, num_periods * sizeof(float)));
    gpuErrchk(hipMalloc(&dev_period_dts, num_p_dts * sizeof(float)));
    gpuErrchk(hipMemcpy(dev_periods, periods, num_periods * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dev_period_dts, period_dts, num_p_dts * sizeof(float), hipMemcpyHostToDevice));

    // Intermediate conditional entropy memory
    float* dev_per;
    gpuErrchk(hipMalloc(&dev_per, num_curves * per_out_size));

    // Kernel launch information
    const size_t x_threads = 256;
    const size_t y_threads = 1;
    const size_t x_blocks = ((num_periods + x_threads - 1) / x_threads);
    const size_t y_blocks = ((num_p_dts + y_threads - 1) / y_threads);
    const dim3 block_dim = dim3(x_threads, y_threads);
    const dim3 grid_dim = dim3(x_blocks, y_blocks);

    // Buffer size (large enough for the longest light curve)
    auto max_length = std::max_element(lengths.begin(), lengths.end());
    const size_t buffer_length = *max_length;
    const size_t buffer_bytes = sizeof(float) * buffer_length * num_curves;
    printf("num_curves: %lu\n", num_curves);
    printf("buffer length (max_length): %lu\n", buffer_length);
    printf("buffer_bytes: %lu\n", buffer_bytes);
    printf("lengths.size(), bytes: %lu\t%lu\n", lengths.size(), lengths.size() * sizeof(size_t));
    printf("times.size(), bytes: %lu\t%lu\n", times.size(), times.size() * sizeof(float*));
    printf("mags.size(), bytes: %lu\t%lu\n", mags.size(), mags.size() * sizeof(float*));
    printf("per_out_size: %lu\n", per_out_size);

    // Allocate for lengths on the device
    size_t* dev_lengths;
    gpuErrchk(hipMalloc(&dev_lengths, num_curves * sizeof(size_t)));

    float* dev_times_buffer;
    float* dev_mags_buffer;
    gpuErrchk(hipMalloc(&dev_times_buffer, buffer_bytes));
    gpuErrchk(hipMalloc(&dev_mags_buffer, buffer_bytes));

    // Zero conditional entropy output
    gpuErrchk(hipMemset(dev_per, 0, per_out_size));

    for (size_t batch_idx = 0; batch_idx < lengths.size(); batch_idx += num_curves) {
        size_t curve_bytes = 0;
        // Copy light curve into device buffer
        for (size_t i = 0; i < num_curves; i++) {
            if (batch_idx + i < lengths.size()) {
                curve_bytes += lengths[batch_idx + i] * sizeof(float);
            }
        }

        //printf("HOST: Copying %lu bytes for curve %lu\n", curve_bytes, batch_idx);
        gpuErrchk(hipMemcpy(dev_times_buffer, times[batch_idx], curve_bytes, hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(dev_mags_buffer, mags[batch_idx], curve_bytes, hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(dev_lengths, &lengths[batch_idx], num_curves * sizeof(size_t), hipMemcpyHostToDevice));

        LombScargleKernelBatched<<<grid_dim, block_dim>>>(
            dev_times_buffer, dev_mags_buffer, dev_lengths,
            dev_periods, dev_period_dts, num_periods, num_p_dts, num_curves,
            buffer_length, *this, dev_per);

        gpuErrchk(hipPeekAtLastError());  // Check for kernel launch errors
        gpuErrchk(hipDeviceSynchronize());  // Ensure kernel execution completes

        // Copy periodogram back to host
        gpuErrchk(hipMemcpy(&per_out[batch_idx * per_points], dev_per, num_curves * per_out_size, hipMemcpyDeviceToHost));
    }

    // Free all of the GPU memory
    gpuErrchk(hipFree(dev_periods));
    gpuErrchk(hipFree(dev_period_dts));
    gpuErrchk(hipFree(dev_lengths));  // Free dev_lengths
    gpuErrchk(hipFree(dev_per));
    gpuErrchk(hipFree(dev_times_buffer));
    gpuErrchk(hipFree(dev_mags_buffer));
}



float* LombScargle::CalcLSBatched(const std::vector<float*>& times,
                                  const std::vector<float*>& mags,
                                  const std::vector<size_t>& lengths,
                                  const float* periods,
                                  const float* period_dts,
                                  const size_t num_periods,
                                  const size_t num_p_dts) const {
    // Size of one periodogram out array, and total periodogram output size.
    size_t per_points = num_periods * num_p_dts;
    size_t per_out_size = per_points * sizeof(float);
    size_t per_size_total = per_out_size * lengths.size();

    // Allocate the output CE array so we can copy to it.
    float* per_out = (float*)malloc(per_size_total);

    CalcLSBatched(times, mags, lengths, periods, period_dts, num_periods,
                  num_p_dts, per_out);

    return per_out;
}