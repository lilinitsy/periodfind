#include "hip/hip_runtime.h"
// Copyright 2020 California Institute of Technology. All rights reserved.
// Use of this source code is governed by a BSD-style license that can be
// found in the LICENSE file.
// Author: Ethan Jaszewski

#include "ls.h"

#include <algorithm>

#include <cstdio>

#include "hip/hip_runtime.h"
#include "math.h"

#include "errchk.cuh"

const float TWO_PI = M_PI * 2.0;

//
// Simple LombScargle Function Definitions
//

LombScargle::LombScargle() {}

//
// CUDA Kernels
//

__global__ void LombScargleKernel(const float* times,
                                  const float* mags,
                                  const size_t length,
                                  const float* periods,
                                  const float* period_dts,
                                  const size_t num_periods,
                                  const size_t num_period_dts,
                                  float* periodogram) {
    const size_t thread_x = threadIdx.x + blockIdx.x * blockDim.x;
    const size_t thread_y = threadIdx.y + blockIdx.y * blockDim.y;

    if (thread_x >= num_periods || thread_y >= num_period_dts) {
        return;
    }

    // Period and period time derivative
    const float period = periods[thread_x];
    const float period_dt = period_dts[thread_y];

    // Time derivative correction factor.
    const float pdt_corr = (period_dt / period) / 2;

    float mag_cos = 0.0;
    float mag_sin = 0.0;
    float cos_cos = 0.0;
    float cos_sin = 0.0;

    float cos, sin, i_part;

    for (size_t idx = 0; idx < length; idx++) {
        float t = times[idx];
        float mag = mags[idx];

        float t_corr = t - pdt_corr * t * t;
        float folded = fabsf(modff(t_corr / period, &i_part));

        sincosf(TWO_PI * folded, &sin, &cos);

        mag_cos += mag * cos;
        mag_sin += mag * sin;
        cos_cos += cos * cos;
        cos_sin += cos * sin;
    }

    float sin_sin = static_cast<float>(length) - cos_cos;

    float cos_tau, sin_tau;
    sincosf(0.5 * atan2f(2.0 * cos_sin, cos_cos - sin_sin), &sin_tau, &cos_tau);

    float numerator_l = cos_tau * mag_cos + sin_tau * mag_sin;
    numerator_l *= numerator_l;

    float numerator_r = cos_tau * mag_sin - sin_tau * mag_cos;
    numerator_r *= numerator_r;

    float denominator_l = cos_tau * cos_tau * cos_cos
                          + 2 * cos_tau * sin_tau * cos_sin
                          + sin_tau * sin_tau * sin_sin;

    float denominator_r = cos_tau * cos_tau * sin_sin
                          - 2 * cos_tau * sin_tau * cos_sin
                          + sin_tau * sin_tau * cos_cos;

    periodogram[thread_x * num_period_dts + thread_y] =
        0.5 * ((numerator_l / denominator_l) + (numerator_r / denominator_r));
}

__global__ void LombScargleKernelBatched(const float* times,
                                         const float* mags,
                                         const size_t* lengths,
                                         const float* periods,
                                         const float* period_dts,
                                         const size_t num_periods,
                                         const size_t num_period_dts,
                                         const size_t num_curves,
                                         float* periodogram) {
    const size_t thread_x = threadIdx.x + blockIdx.x * blockDim.x;
    const size_t thread_y = threadIdx.y + blockIdx.y * blockDim.y;

    for (size_t curve_idx = 0; curve_idx < num_curves; curve_idx++) {
        if (thread_x >= num_periods || thread_y >= num_period_dts) {
            return;
        }


        const size_t length = lengths[curve_idx];
        //printf("(curve_idx, length): (%lu, %lu)\n", curve_idx, length);
        //printf("length: %lu\n", length);
        size_t offset = 0;
        for(size_t i = 0; i < curve_idx; i++) // maybe this should start at 1 so it doesn't offset for the first one?
        {
            offset += lengths[i];
        }

        // Period and period time derivative
        const float period = periods[thread_x];
        const float period_dt = period_dts[thread_y];

        // Time derivative correction factor.
        const float pdt_corr = (period_dt / period) / 2;

        float mag_cos = 0.0;
        float mag_sin = 0.0;
        float cos_cos = 0.0;
        float cos_sin = 0.0;

        float cos, sin, i_part;

        for (size_t idx = 0; idx < length; idx++) {
            float t = times[idx + offset];
            float mag = mags[idx + offset];

            float t_corr = t - pdt_corr * t * t;
            float folded = fabsf(modff(t_corr / period, &i_part));

            sincosf(TWO_PI * folded, &sin, &cos);

            mag_cos += mag * cos;
            mag_sin += mag * sin;
            cos_cos += cos * cos;
            cos_sin += cos * sin;
        }

        float sin_sin = static_cast<float>(length) - cos_cos;

        float cos_tau, sin_tau;
        sincosf(0.5 * atan2f(2.0 * cos_sin, cos_cos - sin_sin), &sin_tau,
                &cos_tau);

        float numerator_l = cos_tau * mag_cos + sin_tau * mag_sin;
        numerator_l *= numerator_l;

        float numerator_r = cos_tau * mag_sin - sin_tau * mag_cos;
        numerator_r *= numerator_r;

        float denominator_l = cos_tau * cos_tau * cos_cos
                              + 2 * cos_tau * sin_tau * cos_sin
                              + sin_tau * sin_tau * sin_sin;

        float denominator_r = cos_tau * cos_tau * sin_sin
                              - 2 * cos_tau * sin_tau * cos_sin
                              + sin_tau * sin_tau * cos_cos;

        periodogram[(curve_idx * num_periods * num_period_dts)
                    + (thread_x * num_period_dts + thread_y)] =
            0.5
            * ((numerator_l / denominator_l) + (numerator_r / denominator_r));
    }
}

//
// Wrapper Functions
//

float* LombScargle::DeviceCalcLS(const float* times,
                                 const float* mags,
                                 const size_t length,
                                 const float* periods,
                                 const float* period_dts,
                                 const size_t num_periods,
                                 const size_t num_p_dts) const {
    float* periodogram;
    gpuErrchk(
        hipMalloc(&periodogram, num_periods * num_p_dts * sizeof(float)));

    const size_t x_threads = 256;
    const size_t y_threads = 1;
    const size_t x_blocks = ((num_periods + x_threads - 1) / x_threads);
    const size_t y_blocks = ((num_p_dts + y_threads - 1) / y_threads);

    const dim3 block_dim = dim3(x_threads, y_threads);
    const dim3 grid_dim = dim3(x_blocks, y_blocks);

    LombScargleKernel<<<grid_dim, block_dim>>>(times, mags, length, periods,
                                               period_dts, num_periods,
                                               num_p_dts, periodogram);

    return periodogram;
}

void LombScargle::CalcLS(float* times,
                         float* mags,
                         size_t length,
                         const float* periods,
                         const float* period_dts,
                         const size_t num_periods,
                         const size_t num_p_dts,
                         float* per_out) const {
    CalcLSBatched(std::vector<float*>{times}, std::vector<float*>{mags},
                  std::vector<size_t>{length}, periods, period_dts, num_periods,
                  num_p_dts, per_out);
}

float* LombScargle::CalcLS(float* times,
                           float* mags,
                           size_t length,
                           const float* periods,
                           const float* period_dts,
                           const size_t num_periods,
                           const size_t num_p_dts) const {
    return CalcLSBatched(std::vector<float*>{times}, std::vector<float*>{mags},
                         std::vector<size_t>{length}, periods, period_dts,
                         num_periods, num_p_dts);
}

void LombScargle::CalcLSBatched(const std::vector<float*>& times,
                                const std::vector<float*>& mags,
                                const std::vector<size_t>& lengths,
                                const float* periods,
                                const float* period_dts,
                                const size_t num_periods,
                                const size_t num_p_dts,
                                float* per_out) const {
    // TODO: Use async memory transferring
    // TODO: Look at ways of batching data transfer.

    // Size of one periodogram out array, and total periodogram output size.
    const size_t num_curves = 2;
    size_t per_points = num_periods * num_p_dts;
    size_t per_out_size = num_curves * per_points * sizeof(float);
    size_t per_size_total = per_out_size * lengths.size();

    // Copy trial information over
    float* dev_periods;
    float* dev_period_dts;
    gpuErrchk(hipMalloc(&dev_periods, num_periods * sizeof(float)));
    gpuErrchk(hipMalloc(&dev_period_dts, num_p_dts * sizeof(float)));
    gpuErrchk(hipMemcpy(dev_periods, periods, num_periods * sizeof(float),
                         hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dev_period_dts, period_dts, num_p_dts * sizeof(float),
                         hipMemcpyHostToDevice));

    // Intermediate conditional entropy memory
    float* dev_per;
    gpuErrchk(hipMalloc(&dev_per, per_out_size));

    // Kernel launch information
    const size_t x_threads = 256;
    const size_t y_threads = 1;
    const size_t x_blocks = ((num_periods + x_threads - 1) / x_threads);
    const size_t y_blocks = ((num_p_dts + y_threads - 1) / y_threads);
    const dim3 block_dim = dim3(x_threads, y_threads);
    const dim3 grid_dim = dim3(x_blocks, y_blocks);

    // Buffer size (large enough for longest light curve)
    auto max_length = std::max_element(lengths.begin(), lengths.end());
    const size_t buffer_length = *max_length;
    const size_t buffer_bytes = num_curves * buffer_length * sizeof(float);

    float* dev_times_buffer;
    float* dev_mags_buffer;
    size_t* dev_lengths_buffer;
    gpuErrchk(hipMalloc(&dev_times_buffer, buffer_bytes));
    gpuErrchk(hipMalloc(&dev_mags_buffer, buffer_bytes));
    gpuErrchk(hipMalloc(&dev_lengths_buffer, num_curves * sizeof(size_t)));
    

    printf("num_curves: %lu\n", num_curves);
    printf("buffer length (max_length): %lu\n", buffer_length);
    printf("buffer_bytes: %lu\n", buffer_bytes);
    printf("lengths.size(), bytes: %lu\t%lu\n", lengths.size(), lengths.size() * sizeof(size_t));
    printf("times.size(), bytes: %lu\t%lu\n", times.size(), times.size() * sizeof(float*));
    printf("mags.size(), bytes: %lu\t%lu\n", mags.size(), mags.size() * sizeof(float*));
    printf("per_out_size: %lu\n", per_out_size);
    printf("per_size_total: %lu bytes = %lu Kb = %lu Mb\n", per_size_total, per_size_total / 1024, per_size_total / (1024 * 1024));
    printf("num_periods: %lu\n", num_periods);
    printf("num_p_dts: %lu\n", num_p_dts);
    printf("per_points: %lu\n", per_points);

    for (size_t batch_idx = 0; batch_idx < lengths.size(); batch_idx += num_curves) {
        // Copy light curve into device buffer
        //size_t curve_bytes = 0;
        //const size_t curve_bytes = lengths[batch_idx] * sizeof(float);
        size_t curve_offset = 0;
        for(size_t i = 0; i < num_curves; i++)
        {
            size_t bytes = lengths[batch_idx + i] * sizeof(float);
            hipMemcpy(dev_times_buffer + curve_offset, times[batch_idx + i], bytes, hipMemcpyHostToDevice);
            hipMemcpy(dev_mags_buffer + curve_offset, mags[batch_idx + i], bytes, hipMemcpyHostToDevice);
            curve_offset += lengths[batch_idx + i];
        }
        
        /*hipMemcpy(dev_times_buffer, times[batch_idx], curve_bytes,
                   hipMemcpyHostToDevice);
        hipMemcpy(dev_mags_buffer, mags[batch_idx], curve_bytes,
                   hipMemcpyHostToDevice);*/
        hipMemcpy(dev_lengths_buffer, &lengths[batch_idx], num_curves * sizeof(size_t), hipMemcpyHostToDevice);

        // Zero conditional entropy output
        //gpuErrchk(hipMemset(dev_per, 0, per_out_size));

        LombScargleKernelBatched<<<grid_dim, block_dim>>>(
            dev_times_buffer, dev_mags_buffer, dev_lengths_buffer, dev_periods,
            dev_period_dts, num_periods, num_p_dts, num_curves, dev_per);

        // Copy periodogram back to host
        hipMemcpy(&per_out[batch_idx * per_points], dev_per, per_out_size,
                   hipMemcpyDeviceToHost);
    }

    // Free all of the GPU memory
    gpuErrchk(hipFree(dev_periods));
    gpuErrchk(hipFree(dev_period_dts));
    gpuErrchk(hipFree(dev_per));
    gpuErrchk(hipFree(dev_lengths_buffer));
    gpuErrchk(hipFree(dev_times_buffer));
    gpuErrchk(hipFree(dev_mags_buffer));
}

float* LombScargle::CalcLSBatched(const std::vector<float*>& times,
                                  const std::vector<float*>& mags,
                                  const std::vector<size_t>& lengths,
                                  const float* periods,
                                  const float* period_dts,
                                  const size_t num_periods,
                                  const size_t num_p_dts) const {
    // Size of one periodogram out array, and total periodogram output size.
    size_t per_points = num_periods * num_p_dts;
    size_t per_out_size = per_points * sizeof(float);
    size_t per_size_total = per_out_size * lengths.size();

    // Allocate the output CE array so we can copy to it.
    float* per_out = (float*)malloc(per_size_total);

    CalcLSBatched(times, mags, lengths, periods, period_dts, num_periods,
                  num_p_dts, per_out);

    return per_out;
}